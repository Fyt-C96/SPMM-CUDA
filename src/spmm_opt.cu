#include "hip/hip_runtime.h"
#include "spmm_opt.h"
#include <stdio.h>
__global__ void spmm_kernel_opt(int *ptr, int *idx, float *val, float *vin, float *vout, int num_v, int feat_in)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_v)
        return;
    int begin = ptr[tid], end = ptr[tid + 1];
    for (int j = 0; j < feat_in; ++j)
    {
        float result = 0.0f;
        for (int i = begin; i < end; ++i)
        {
            // Transposing the vin maybe cache-friendly
            result += vin[idx[i] * feat_in + j] * val[i];
        }
        vout[tid * feat_in + j] = result;
    }
}

void SpMMOpt::preprocess(float *vin, float *vout)
{
    // dbg("TODO");
    int BLOCK_SIZE = 1024;
    grid.x = (num_v + BLOCK_SIZE - 1) / BLOCK_SIZE;
    block.x = BLOCK_SIZE;
}

void SpMMOpt::run(float *vin, float *vout)
{
    // dbg("TODO");
    // printf("Grid = <%d, %d, %d>\n", grid.x, grid.y, grid.z);
    // printf("Block = <%d, %d, %d>\n", block.x, block.y, block.z);
    spmm_kernel_opt<<<grid, block>>>(d_ptr, d_idx, d_val, vin, vout, num_v, feat_in);
}