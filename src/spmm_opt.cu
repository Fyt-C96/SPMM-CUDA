#include "hip/hip_runtime.h"
#include "spmm_opt.h"
#include <stdio.h>
__global__ void spmm_kernel_opt(int *ptr, int *idx, float *val, float *vin, float *vout, int num_v, int INFEATURE)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_v)
        return;
    int begin = ptr[tid], end = ptr[tid + 1];
    for (int j = 0; j < INFEATURE; ++j)
    {
        float result = 0.0f;
        for (int i = begin; i < end; ++i)
        {
            result += vin[idx[i] * INFEATURE + j] * val[i];
        }
        vout[tid * INFEATURE + j] = result;
    }
}

void SpMMOpt::preprocess(float *vin, float *vout)
{
    // dbg("TODO");
    int BLOCK_SIZE = 1024;
    grid.x = (num_v + BLOCK_SIZE - 1) / BLOCK_SIZE;
    block.x = BLOCK_SIZE;
}

void SpMMOpt::run(float *vin, float *vout)
{
    // dbg("TODO");
    printf("Grid = <%d, %d, %d>\n", grid.x, grid.y, grid.z);
    printf("Block = <%d, %d, %d>\n", block.x, block.y, block.z);
    spmm_kernel_opt<<<grid, block>>>(d_ptr, d_idx, d_val, vin, vout, num_v, feat_in);
}