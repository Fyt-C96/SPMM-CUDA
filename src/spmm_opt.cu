#include "hip/hip_runtime.h"
#include "spmm_opt.h"
#include <stdio.h>

__global__ void spmm_kernel_opt(int *ptr, int *idx, float *val, float *vin, float *vout, int num_v, int feat_in)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_v)
        return;
    int begin = ptr[tid], end = ptr[tid + 1];
    for (int j = 0; j < feat_in; ++j)
    {
        float result = 0.0f;
        for (int i = begin; i < end; ++i)
        {
            // Transposing the vin maybe cache-friendly
            result += vin[idx[i] + j * num_v] * val[i];
        }
        vout[tid * feat_in + j] = result;
    }

}

void SpMMOpt::preprocess(float *vin, float *vout)
{
    // dbg("TODO");
    int BLOCK_SIZE = 1024;
    grid.x = (num_v + BLOCK_SIZE - 1) / BLOCK_SIZE;
    block.x = BLOCK_SIZE;
}

void SpMMOpt::run(float *vin, float *vout)
{
    // dbg("TODO");
    // printf("Grid = <%d, %d, %d>\n", grid.x, grid.y, grid.z);
    // printf("Block = <%d, %d, %d>\n", block.x, block.y, block.z);
    float *new_vin;
    hipMalloc(&new_vin, feat_in * num_v * sizeof(float));
    hipblasCreate(&handle);
    float alpha = 1, beta = 0;
    hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, num_v, feat_in, &alpha, vin, feat_in,
                        &beta,
                        nullptr, num_v,
                        new_vin, num_v);
    spmm_kernel_opt<<<grid, block>>>(d_ptr, d_idx, d_val, new_vin, vout, num_v, feat_in);
}